#include "ParticleSwarmOptimizer.cuh"
#include <thrust/iterator/counting_iterator.h>
#include <thrust/extrema.h>
#include <thrust/for_each.h>
#include <hiprand/hiprand_kernel.h>

namespace INTERNAL
{
	namespace PSO
	{
		InitFunctor::InitFunctor(int dimSize, double (*function)(double*, int)) :
			m_dimSize(dimSize),
			m_function(function)
		{}

		__device__
			void InitFunctor::operator()(const thrust::tuple<Particle&, int>& particleIndexTuple) const
		{
			Particle& particle = particleIndexTuple.get<0>();
			int index = particleIndexTuple.get<1>();

			hiprandState_t randomState;
			hiprand_init(index, 0, 0, &randomState);

			particle = Particle(m_dimSize, m_function, randomState);
		}

		RunFunctor::RunFunctor(double aInd, double aGrp, double w, double* currentGlobalBest, double* boundsVector, double* boundsSize) :
			m_aInd(aInd),
			m_aGrp(aGrp),
			m_w(w),
			m_currentGlobalBest(currentGlobalBest),
			m_boundsVector(boundsVector),
			m_boundsSize(boundsSize)
		{}

		__device__
			void RunFunctor::operator()(const thrust::tuple<Particle&, double*&>& particlePersonalBestTuple)
		{
			Particle& particle = particlePersonalBestTuple.get<0>();
			double* personalBest = particlePersonalBestTuple.get<1>();
			particle.Run(m_aInd, m_aGrp, m_w, personalBest, m_currentGlobalBest, m_boundsVector, m_boundsSize);
		}
	}
}

ParticleSwarmOptimizer::ParticleSwarmOptimizer(int numParticles, double (*function)(double*, int), int numDimensions)
{
	thrust::counting_iterator<int> counter = thrust::make_counting_iterator(0);
	auto beginIterator = thrust::make_zip_iterator(thrust::make_tuple(mdv_particles.begin(), counter));
	auto endIterator = thrust::make_zip_iterator(thrust::make_tuple(mdv_particles.end(), counter + numDimensions));
	INTERNAL::PSO::InitFunctor initFunctor(numDimensions, function);
	thrust::for_each(beginIterator, endIterator, initFunctor);
}

void ParticleSwarmOptimizer::Run(int numIterations, double aInd, double aGrp, double w, const thrust::host_vector<double>& boundsVector, const thrust::host_vector<double>& boundsSize)
{
	mdv_boundsVector = boundsVector;
	mdv_boundsSize = boundsSize;

	auto beginIterator = thrust::make_zip_iterator(thrust::make_tuple(mdv_particles.begin(), currentParticleBestPositionsPointers.begin()));
	auto endIterator = thrust::make_zip_iterator(thrust::make_tuple(mdv_particles.end(), currentParticleBestPositionsPointers.end()));
	INTERNAL::PSO::RunFunctor runFunctor(aInd, aGrp, w, mdv_currentGlobalBestPosition.data().get(), mdv_boundsVector.data().get(), mdv_boundsSize.data().get());

	for (int i = 0; i < numIterations; ++i)
	{
		thrust::for_each(beginIterator, endIterator, runFunctor);
		auto minElementIter = thrust::min_element(mdv_particles.begin(), mdv_particles.end(), [] __device__(const Particle & a, const Particle & b) { return a.GetMinimum() < b.GetMinimum(); });
		auto minElementIndex = minElementIter - mdv_particles.begin();


		mdv_currentGlobalBestPosition = mdv_currentParticleBestPositions[minElementIndex];
	}
}
