#include "Particle.cuh"

__device__
Particle::Particle(int dimSize, double (*function)(double*, int), const hiprandState_t& randomState) :
	m_dimSize(dimSize),
	m_function(function),
	m_randState(randomState),
	m_currentVelocity(new double[dimSize]),
	m_currentPosition(new double[dimSize])
{
	memset(m_currentVelocity, 0, m_dimSize * sizeof(double));
}

__device__
Particle::Particle()
{}

__device__
void Particle::Run(double aInd, double aGrp, double w, double* currentPersonalBest, const double* currentGlobalBest, const double* boundsVector, const double* boundsSize)
{

	//Initialize random location
	for (int i = 0; i < m_dimSize; ++i) { m_currentPosition[i] = boundsVector[i] + boundsSize[i] * hiprand_uniform_double(&m_randState); }

	for (int i = 0; i < m_dimSize; ++i)
	{
		double rInd = hiprand_uniform_double(&m_randState);
		double rGrp = hiprand_uniform_double(&m_randState);

		m_currentVelocity[i] = w * m_currentVelocity[i] + aInd * rInd * (currentPersonalBest[i] - m_currentPosition[i]) + aGrp * rGrp * (currentGlobalBest[i] - m_currentPosition[i]);
		m_currentPosition[i] = m_currentPosition[i] * m_currentVelocity[i];
	}
	auto currentValue = m_function(m_currentPosition, m_dimSize);
	if (currentValue < m_currentMinimum)
	{
		m_currentMinimum = currentValue;
		for (int i = 0; i < m_dimSize; ++i) { currentPersonalBest[i] = m_currentPosition[i]; }
	}
}

__device__
double Particle::GetMinimum() const { return m_currentMinimum; }

__device__
Particle::~Particle()
{
	delete[] m_currentVelocity;
	delete[] m_currentPosition;
}