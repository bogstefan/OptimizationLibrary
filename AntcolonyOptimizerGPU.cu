#include "hip/hip_runtime.h"
#include "AntcolonyOptimizerGPU.cuh"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <thrust/extrema.h>
#include <time.h>
#include <iostream>


#ifdef __INTELLISENSE__
double atomicAdd(double* address, double val) {};
#endif

//using namespace SIMPLE;


namespace INTERNAL {
	namespace PSO 
	{

		__device__
			void InitFunctor::operator()(const thrust::tuple<AntGPU&, int>& antIndexTuple) const
		{
			AntGPU& ant = antIndexTuple.get<0>();
			int index = antIndexTuple.get<1>();
			hiprandState_t randomState;
			hiprand_init(clock() + index, 0, 0, &randomState);
			//hiprand_init(clock(), index, 0, &randomState);
			ant = AntGPU(initialLocation, matrixDim, randomState);
		}

		InitFunctor::InitFunctor(int initialLocation, int matrixDim) :
			initialLocation(initialLocation),
			matrixDim(matrixDim)
		{}

		__device__
			void VentureFunctor::operator()(const thrust::tuple<AntGPU&, int*&>& antRouteTuple) const
		{
			AntGPU& ant = antRouteTuple.get<0>();
			int* route = antRouteTuple.get<1>();

			ant.Reset(route, initialLocation, matrixDim);
			ant.Venture(route, distanceMatrix, pheromoneMatrix, matrixDim, alpha, beta);
		}

		VentureFunctor::VentureFunctor(double* distanceMatrix, double* pheromoneMatrix, int matrixDim,
			int initialLocation, double alpha, double beta) :
			distanceMatrix(distanceMatrix),
			pheromoneMatrix(pheromoneMatrix),
			matrixDim(matrixDim),
			initialLocation(initialLocation),
			alpha(alpha),
			beta(beta)
		{}

		__device__
			void SetPheromoneChangesFunctor::operator()(const thrust::tuple<AntGPU&, int*&>& antRouteTuple) const
		{
			AntGPU& ant = antRouteTuple.get<0>();
			int* route = antRouteTuple.get<1>();

			for (int i = 0; i <= matrixDim - 1; ++i)
			{
				int currentNode = route[i];
				int nextNode = route[i + 1];
				double pheromoneAddition = 1.0 / ant.distance;

				atomicAdd(&pheromoneMatrixChanges[matrixDim * currentNode + nextNode], pheromoneAddition);
				atomicAdd(&pheromoneMatrixChanges[matrixDim * nextNode + currentNode], pheromoneAddition);
			}
		}

		SetPheromoneChangesFunctor::SetPheromoneChangesFunctor(double* pheromoneMatrixChanges, int matrixDim) :
			pheromoneMatrixChanges(pheromoneMatrixChanges),
			matrixDim(matrixDim)
		{}

		__device__
			void EvaporatePheromoneFunctor::operator()(double& x) const
		{
			x = (1.0 - evaporationRate) * x;
		}

		EvaporatePheromoneFunctor::EvaporatePheromoneFunctor(double evaporationRate) :
			evaporationRate(evaporationRate)
		{}

		void UpdatePheromonesFunctor::operator()(const thrust::tuple<double&, double&>& pheromonePheromoneChangesTuple) const
		{
			double& pheromone = pheromonePheromoneChangesTuple.get<0>();
			double pheromoneChange = pheromonePheromoneChangesTuple.get<1>();
			pheromone += pheromoneChange;
		}

	}
}

AntcolonyOptimizerGPU::AntcolonyOptimizerGPU(int numAnts, int initialLocation, thrust::host_vector<double> distanceMatrix, thrust::host_vector<double> pheromoneMatrix, int matrixDim) :
	mh_numAnts(numAnts),
	m_matrixDim(matrixDim),
	mdv_ants(numAnts),
	mdv_distanceMatrix(distanceMatrix),
	mdv_pheromoneMatrix(pheromoneMatrix),
	mdv_pheromoneMatrixChanges(matrixDim* matrixDim, 0.0),
	mh_bestRoute(matrixDim),
	mdv_routes(numAnts, thrust::device_vector<int>(matrixDim + 1, 9)),
	m_initialLocation(initialLocation)
{
	m_bestDistance = 999999999;
	thrust::host_vector<int*> deviceRoutePointersH(mh_numAnts);

	for (int i = 0; i < mh_numAnts; ++i)
	{
		deviceRoutePointersH[i] = mdv_routes[i].data().get();
	}

	mdv_routePointers = deviceRoutePointersH;

	thrust::counting_iterator<int> counter = thrust::make_counting_iterator(0);

	auto start = thrust::make_zip_iterator(thrust::make_tuple(mdv_ants.begin(), counter));
	auto end = thrust::make_zip_iterator(thrust::make_tuple(mdv_ants.end(), counter + numAnts));

	INTERNAL::PSO::InitFunctor initFunctor(initialLocation, matrixDim);
	thrust::for_each(start, end, initFunctor);
}


void AntcolonyOptimizerGPU::Run(int numIterations, double alpha, double beta, double evaporationRate)
{
	auto antRouteBegin = thrust::make_zip_iterator(thrust::make_tuple(mdv_ants.begin(), mdv_routePointers.begin()));
	auto antRouteEnd = thrust::make_zip_iterator(thrust::make_tuple(mdv_ants.end(), mdv_routePointers.end()));

	INTERNAL::PSO::VentureFunctor ventureFunctor(mdv_distanceMatrix.data().get(), mdv_pheromoneMatrix.data().get(), m_matrixDim, m_initialLocation, alpha, beta);
	INTERNAL::PSO::SetPheromoneChangesFunctor setPheromoneChangesFunctor(mdv_pheromoneMatrixChanges.data().get(), m_matrixDim);

	INTERNAL::PSO::EvaporatePheromoneFunctor evaporatePheromoneFunctor(evaporationRate);

	INTERNAL::PSO::UpdatePheromonesFunctor updatePheromonesFunctor;
	auto pheromonePheromoneChangesBegin = thrust::make_zip_iterator(thrust::make_tuple(mdv_pheromoneMatrix.begin(), mdv_pheromoneMatrixChanges.begin()));
	auto pheromonePheromoneChangesEnd = thrust::make_zip_iterator(thrust::make_tuple(mdv_pheromoneMatrix.end(), mdv_pheromoneMatrixChanges.end()));

	for (int i = 0; i < numIterations; ++i)
	{
		std::cout << "Iteration Nr: " << i << '\n';

		std::cout << "Venturing\n";
		thrust::for_each(antRouteBegin, antRouteEnd, ventureFunctor);

		std::cout << "Set Pheromone Delta\n";
		thrust::for_each(antRouteBegin, antRouteEnd, setPheromoneChangesFunctor);
	
		thrust::for_each(mdv_pheromoneMatrix.begin(), mdv_pheromoneMatrix.begin() + m_matrixDim * m_matrixDim, evaporatePheromoneFunctor);

		//thrust::for_each(pheromonePheromoneChangesBegin, pheromonePheromoneChangesEnd, updatePheromonesFunctor);
		thrust::transform(mdv_pheromoneMatrix.begin(), mdv_pheromoneMatrix.end(), mdv_pheromoneMatrixChanges.begin(), mdv_pheromoneMatrix.begin(), thrust::plus<double>());
		//thrust::host_vector<double> after = mdv_pheromoneMatrix;

		auto minDistanceIter = thrust::min_element(mdv_ants.begin(), mdv_ants.end(), [] __device__(const AntGPU & lhs, const AntGPU & rhs)
		{
			return lhs.distance < rhs.distance;
		});

		int minDistanceIndex = minDistanceIter - mdv_ants.begin();
		AntGPU minDistanceAnt = *minDistanceIter;
		double minDistance = minDistanceAnt.distance;


		if (minDistance < m_bestDistance)
		{
			m_bestDistance = minDistance;
			mh_bestRoute = mdv_routes[minDistanceIndex];
		}

		thrust::fill(thrust::device, mdv_pheromoneMatrixChanges.begin(), mdv_pheromoneMatrixChanges.end(), 0.0);
	}
}

void AntcolonyOptimizerGPU::Report()
{
	std::cout << "The Optimal Length is: " << m_bestDistance << std::endl;
}
