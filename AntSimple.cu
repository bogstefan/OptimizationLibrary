#include "hip/hip_runtime.h"
#include "AntSimple.cuh"
#include <stdio.h>

namespace SIMPLE
{
	__device__
		Ant::Ant(int initialLocation, int matrixDim, hiprandState_t randState) :
		visitedIndex(0),
		isVisited(new bool[matrixDim]),
		position(initialLocation),
		goodnessNumerators(new double[matrixDim]),
		m_randomState(randState)
	{
	}

	__device__
		void Ant::Venture(int* route, const double* distanceMatrix, const double* pheromoneMatrix, int matrixDim, double alpha, double beta)
	{
		while (visitedIndex < matrixDim)
		{
			int nextHop = SelectNextHop(distanceMatrix, pheromoneMatrix, matrixDim, alpha, beta);
			GoTo(nextHop, route, distanceMatrix, matrixDim);
		}
		route[matrixDim] = route[0];
		distance += distanceMatrix[route[matrixDim - 1] * matrixDim + route[0]];
		//printf("Distance Traveled: %f\n", distance);
	}

	__device__
		int Ant::SelectNextHop(const double* distance_matrix, const double* pheromoneMatrix, int matrixDim, double alpha, double beta)
	{
		double denominator = 0;
		for (int i = 0; i < matrixDim; ++i)
		{
			if (isVisited[i]) { continue; }
			int possiblePosition = i;

			double goodnessNumerator = pow(pheromoneMatrix[position * matrixDim + possiblePosition], alpha) * pow(1.0 / distance_matrix[position * matrixDim + possiblePosition], beta);

			goodnessNumerators[possiblePosition] = goodnessNumerator;
			denominator += goodnessNumerator;
		}

		//New
		/*
		for (int i = 0; i < matrixDim; ++i)
		{
			if (isVisited[i]) { continue; }
			goodnessNumerators[i] /= denominator;
		}

		double random = hiprand_uniform_double(&m_randomState);

		for (int i = 0; i < matrixDim; ++i)
		{
			if (isVisited[i]) { continue; }
			random -= goodnessNumerators[i];
			if (random <= 0) { return i; }
			
		}
		return -1;
		*/
		

		double sum = 0;
		double random = hiprand_uniform_double(&m_randomState);
		//printf("Random is %f\n", random);
		
		for (int i = 0; i < matrixDim; ++i)
		{
			if (isVisited[i]) { continue; }

			int possiblePosition = i;
			double numerator = goodnessNumerators[possiblePosition];
			double probability = numerator / denominator;
			if (random <= sum + probability)
			{
				return possiblePosition;
			}
			sum += probability;
		}
		return -1;
		
	}

	__device__
		void Ant::GoTo(int next, int* route, const double* distanceMatrix, int matrixDim)
	{
		route[visitedIndex++] = next;
		isVisited[next] = true;
		distance += distanceMatrix[position * matrixDim + next];
		position = next;
	}

	__device__
		void Ant::Reset(int* route, int initialLocation, int matrixDim)
	{
		visitedIndex = 0;
		distance = 0;
		position = initialLocation;
		for (int i = 0; i < matrixDim; ++i) { isVisited[i] = false; }
		isVisited[position] = true;
		route[visitedIndex++] = initialLocation;
	}


}


