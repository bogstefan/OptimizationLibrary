#include "hip/hip_runtime.h"
#include "AntGPU.cuh"
#include <stdio.h>

__device__
AntGPU::AntGPU(int initialLocation, int matrixDim, hiprandState_t randState) :
	position(initialLocation),
	possibleLocations(new int[matrixDim]),
	goodnessNumerators(new double[matrixDim]),
	possibleLocationsLastIndex(matrixDim - 1),
	m_randomState(randState)
{
}

__device__
void AntGPU::Venture(int* route, const double* distanceMatrix, const double* pheromoneMatrix, int matrixDim, double alpha, double beta)
{
	while (possibleLocationsLastIndex >= 0)
	{
		int nextHop = SelectNextHop(distanceMatrix, pheromoneMatrix, matrixDim, alpha, beta);
		GoTo(nextHop, route, distanceMatrix, matrixDim);
	}
	//printf("Distance Traveled: %f\n", distance + distanceMatrix[position]);
	route[matrixDim] = route[0];
	distance += distanceMatrix[route[matrixDim - 1] * matrixDim + route[0]];
}

__device__
int AntGPU::SelectNextHop(const double* distance_matrix, const double* pheromoneMatrix, int matrixDim, double alpha, double beta)
{
	double denominator = 0;
	for (int i = 0; i <= possibleLocationsLastIndex; ++i)
	{
		int possiblePosition = possibleLocations[i];
		double goodnessNumerator = pow(pheromoneMatrix[position * matrixDim + possiblePosition], alpha) * pow(1.0 / distance_matrix[position * matrixDim + possiblePosition], beta);

		goodnessNumerators[possiblePosition] = goodnessNumerator;
		denominator += goodnessNumerator;
	}
	
	double sum = 0;
	double random = hiprand_uniform_double(&m_randomState);

	for (int i = 0; i <= possibleLocationsLastIndex; ++i)
	{
		int possiblePosition = possibleLocations[i];
		double numerator = goodnessNumerators[possiblePosition];
		double probability = numerator / denominator;
		if (random <= sum + probability)
		{
			possibleLocationsNextIndex = i;
			return possiblePosition;
		}
		sum += probability;
	}
	return -1;
	
}

__device__
void AntGPU::GoTo(int next, int* route, const double* distanceMatrix, int matrixDim)
{
	route[routeIndex++] = next;
	possibleLocations[possibleLocationsNextIndex] = possibleLocations[possibleLocationsLastIndex--];
	distance += distanceMatrix[position * matrixDim + next];
	position = next;
}

__device__
void AntGPU::Reset(int* route, int initialLocation, int matrixDim)
{
	routeIndex = 0;
	distance = 0;
	position = initialLocation;
	possibleLocationsLastIndex = matrixDim - 1;
	for (int i = 0; i < matrixDim; ++i) { possibleLocations[i] = i; }
	route[routeIndex++] = initialLocation;
	possibleLocations[initialLocation] = possibleLocations[possibleLocationsLastIndex--];
}


